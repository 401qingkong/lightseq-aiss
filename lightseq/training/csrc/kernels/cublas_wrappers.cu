#include "hip/hip_runtime.h"
/* Copyright 2021 The LightSeq Team
   Copyright Microsoft DeepSpeed
   This file is adapted from Microsoft DeepSpeed
*/
#ifdef __HIPCC__
#include <rocblas.h>
#endif
#include "cublas_wrappers.h"

int cublas_gemm_ex(hipblasHandle_t handle, hipblasOperation_t transa,
                   hipblasOperation_t transb, int m, int n, int k,
                   const float *alpha, const float *beta, const float *A,
                   const float *B, float *C,
#ifdef __HIPCC__
                   rocblas_gemm_algo algo) {
  rocblas_status status = rocblas_gemm_ex(
      handle, transa, transb, m, n, k, (const void *)alpha, (const void *)A,
      rocblas_datatype_f32_r, (transa == rocblas_operation_none) ? m : k,
      (const void *)B, rocblas_datatype_f32_r,
      (transb == rocblas_operation_none) ? k : n, (const void *)beta, C,
      rocblas_datatype_f32_r, m, C, rocblas_datatype_f32_r, m,
      rocblas_datatype_f32_r, algo, 0, 0);

#else
                   hipblasGemmAlgo_t algo) {
  hipblasStatus_t status =
      hipblasGemmEx(handle, transa, transb, m, n, k, (const void *)alpha,
                   (const void *)A, HIP_R_32F, (transa == HIPBLAS_OP_N) ? m : k,
                   (const void *)B, HIP_R_32F, (transb == HIPBLAS_OP_N) ? k : n,
                   (const void *)beta, C, HIP_R_32F, m, HIP_R_32F, algo);

#endif
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr,
            "!!!! kernel execution error. (m: %d, n: %d, k: %d, error: %d) \n",
            m, n, k, (int)status);
    return EXIT_FAILURE;
  }
  return 0;
}

int cublas_gemm_ex(hipblasHandle_t handle, hipblasOperation_t transa,
                   hipblasOperation_t transb, int m, int n, int k,
                   const float *alpha, const float *beta, const __half *A,
                   const __half *B, __half *C,
#ifdef __HIPCC__
                   rocblas_gemm_algo algo) {
  __half alpha_value = __float2half(*alpha);
  __half beta_value = __float2half(*beta);

  rocblas_status status = rocblas_gemm_ex(
      handle, transa, transb, m, n, k, &alpha_value, (const void *)A,
      rocblas_datatype_f16_r, (transa == rocblas_operation_none) ? m : k,
      (const void *)B, rocblas_datatype_f16_r,
      (transb == rocblas_operation_none) ? k : n, &beta_value, (void *)C,
      rocblas_datatype_f16_r, m, (void *)C, rocblas_datatype_f16_r, m,
      rocblas_datatype_f16_r, algo, 0, 0);
#else
                   hipblasGemmAlgo_t algo) {
  hipblasStatus_t status = hipblasGemmEx(
      handle, transa, transb, m, n, k, (const void *)alpha, (const void *)A,
      HIP_R_16F, (transa == HIPBLAS_OP_N) ? m : k, (const void *)B, HIP_R_16F,
      (transb == HIPBLAS_OP_N) ? k : n, (const void *)beta, (void *)C,
      HIP_R_16F, m, HIP_R_32F, algo);
#endif

  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr,
            "!!!! kernel execution error. (m: %d, n: %d, k: %d, error: %d) \n",
            m, n, k, (int)status);
    return EXIT_FAILURE;
  }
  return 0;
}

int cublas_strided_batched_gemm(hipblasHandle_t handle, int m, int n, int k,
                                const float *alpha, const float *beta,
                                const float *A, const float *B, float *C,
                                hipblasOperation_t op_A, hipblasOperation_t op_B,
                                int stride_A, int stride_B, int stride_C,
                                int batch,
#ifdef __HIPCC__
                                rocblas_gemm_algo algo) {
  rocblas_status status = rocblas_gemm_strided_batched_ex(
      handle, op_A, op_B, m, n, k, alpha, A, rocblas_datatype_f32_r,
      (op_A == rocblas_operation_none) ? m : k, stride_A, B,
      rocblas_datatype_f32_r, (op_B == rocblas_operation_none) ? k : n,
      stride_B, beta, C, rocblas_datatype_f32_r, m, stride_C, C,
      rocblas_datatype_f32_r, m, stride_C, batch, rocblas_datatype_f32_r, algo,
      0, 0);
#else
                                hipblasGemmAlgo_t algo) {
  hipblasStatus_t status = hipblasGemmStridedBatchedEx(
      handle, op_A, op_B, m, n, k, alpha, A, HIP_R_32F,
      (op_A == HIPBLAS_OP_N) ? m : k, stride_A, B, HIP_R_32F,
      (op_B == HIPBLAS_OP_N) ? k : n, stride_B, beta, C, HIP_R_32F, m, stride_C,
      batch, HIP_R_32F, algo);
#endif
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr,
            "!!!! kernel execution error. (batch: %d, m: %d, n: %d, k: %d, "
            "error: %d) \n",
            batch, m, n, k, (int)status);
    return EXIT_FAILURE;
  }
  return 0;
}

int cublas_strided_batched_gemm(hipblasHandle_t handle, int m, int n, int k,
                                const float *alpha, const float *beta,
                                const __half *A, const __half *B, __half *C,
                                hipblasOperation_t op_A, hipblasOperation_t op_B,
                                int stride_A, int stride_B, int stride_C,
                                int batch,
#ifdef __HIPCC__
                                rocblas_gemm_algo algo) {
  __half alpha_value = __float2half(*alpha);
  __half beta_value = __float2half(*beta);
  rocblas_status status = rocblas_gemm_strided_batched_ex(
      handle, op_A, op_B, m, n, k, &alpha_value, A, rocblas_datatype_f16_r,
      (op_A == rocblas_operation_none) ? m : k, stride_A, B,
      rocblas_datatype_f16_r, (op_B == rocblas_operation_none) ? k : n,
      stride_B, &beta_value, C, rocblas_datatype_f16_r, m, stride_C, C,
      rocblas_datatype_f16_r, m, stride_C, batch, rocblas_datatype_f16_r, algo,
      0, 0);
#else
                                hipblasGemmAlgo_t algo) {
  hipblasStatus_t status = hipblasGemmStridedBatchedEx(
      handle, op_A, op_B, m, n, k, alpha, A, HIP_R_16F,
      (op_A == HIPBLAS_OP_N) ? m : k, stride_A, B, HIP_R_16F,
      (op_B == HIPBLAS_OP_N) ? k : n, stride_B, beta, C, HIP_R_16F, m, stride_C,
      batch, HIP_R_32F, algo);
#endif
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr,
            "!!!! kernel execution error. (m: %d, n: %d, k: %d, error: %d) \n",
            m, n, k, (int)status);
    return EXIT_FAILURE;
  }

  return 0;
}
